#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "object_dynamics.h"
#include <math.h>

__device__ const double G = 1.610e-3;

__device__ extern __shared__ volatile double sdata[];

/* Taken nearly word for word from http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf */
__device__ void reduce(const unsigned int blockSize, const unsigned int tid)
{
	if (blockSize >= 512) {
		if (tid < 256) { 
			sdata[tid] += sdata[tid + 256]; 
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) { 
			sdata[tid] += sdata[tid + 128]; 
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {  
			sdata[tid] += sdata[tid + 64]; 
		}
		__syncthreads();
	}
	if (tid < 32) {
		if (blockSize >= 64) {
			sdata[tid] += sdata[tid + 32]; 
		}
		if (blockSize >= 32) {
			sdata[tid] += sdata[tid + 16]; 
		}
		if (blockSize >= 16) {
			sdata[tid] += sdata[tid + 8]; 
		}
		if (blockSize >= 8) {
			sdata[tid] += sdata[tid + 4]; 
		}
		if (blockSize >= 4) {
			sdata[tid] += sdata[tid + 2]; 
		}
		if (blockSize >= 2) {
			sdata[tid] += sdata[tid + 1]; 
		}
	}
}

__device__ void kfunction(const unsigned int blockSize, const unsigned int tid, const unsigned int obj, const double3 inpos, double3 &kvel, const double* g_idata, const unsigned int numdynamicobjects,  const unsigned int numdynamicobjects_align) 
{	
	// do object dynamics, each thread may have to do multiple items
	for (int i = tid; i < numdynamicobjects; i+=blockSize) {
		if (i != obj) {
			// read in other object's properties
			double3 o_pos = make_double3(g_idata[numdynamicobjects_align*0 + i],
										g_idata[numdynamicobjects_align*1 + i],
										g_idata[numdynamicobjects_align*2 + i]);
			double o_mass = g_idata[numdynamicobjects_align*6 + i];

			// gravity k > 1
			double3 rvec = make_double3(o_pos.x - inpos.x,
										o_pos.y - inpos.y,
										o_pos.z - inpos.z);

			double rinv = 1.0 / sqrt(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);

			kvel.x += o_mass * rinv*rinv*rinv * rvec.x;
			kvel.y += o_mass * rinv*rinv*rinv * rvec.y;
			kvel.z += o_mass * rinv*rinv*rinv * rvec.z;
		}
	}

	// reduction to get overall k
	sdata[tid] = kvel.x;
	__syncthreads();
	reduce(blockSize, tid);
	__syncthreads();
	kvel.x = sdata[0];

	__syncthreads();
	sdata[tid] = kvel.y;
	__syncthreads();
	reduce(blockSize, tid);
	kvel.y = sdata[0];

	__syncthreads();
	sdata[tid] = kvel.z;
	__syncthreads();
	reduce(blockSize, tid);
	__syncthreads();
	kvel.z = sdata[0];

	__syncthreads();
}

__device__ void cuda_collisions(const unsigned int blockSize, const unsigned int tid, const unsigned int obj, double3 &colvel, const double3 pos, const double3 vel, const double* g_idata, const unsigned int numdynamicobjects,  const unsigned int numdynamicobjects_align)
{
	double mass = g_idata[numdynamicobjects_align*6 + obj],
		   radius = g_idata[numdynamicobjects_align*7 + obj];

	// do object collisions, each thread may have to do multiple items
	for (int i = tid; i < numdynamicobjects; i+=blockSize) {
		if (i != obj) {
			// read in other object's properties
			double3 o_pos = make_double3(g_idata[numdynamicobjects_align*0 + i],
											g_idata[numdynamicobjects_align*1 + i],
											g_idata[numdynamicobjects_align*2 + i]);
			double3 o_vel = make_double3(g_idata[numdynamicobjects_align*3 + i],
											g_idata[numdynamicobjects_align*4 + i],
											g_idata[numdynamicobjects_align*5 + i]);
			double o_mass = g_idata[numdynamicobjects_align*6 + i],
					o_radius = g_idata[numdynamicobjects_align*7 + i];

			double3 Dp = make_double3(o_pos.x - pos.x,
									  o_pos.y - pos.y,
									  o_pos.z - pos.z);
			double d = sqrt(Dp.x*Dp.x + Dp.y*Dp.y + Dp.z*Dp.z);

			if (d <= radius + o_radius){

				// from the other project/game

				double3 u = make_double3(Dp.x/d, Dp.y/d, Dp.z/d);

				// find their speed in the collision direction
				double cs = u.x * vel.x + u.y * vel.y + u.z * vel.z,
					   o_cs = u.x * o_vel.x + u.y * o_vel.y + u.z * o_vel.z;

				// momentum and energy of the objects in the collision direction
				double P = mass*cs + o_mass*o_cs;
				double E = 0.5*(mass*cs*cs + o_mass*o_cs*o_cs);

				// conserve momentum and energy
				double a = mass * ( 1 + mass/o_mass),
						b = -2.0*P*mass/o_mass,
						c = P*P/o_mass - 2.0*E;

				double dcs = (-b - sqrt(b*b - 4.0*a*c)) / (2.0*a);
			
				colvel.x += (dcs - cs)*u.x;
				colvel.y += (dcs - cs)*u.y;
				colvel.z += (dcs - cs)*u.z;
			}
		}
	}
	// reduce
	sdata[tid] = colvel.x;
	__syncthreads();
	reduce(blockSize, tid);
	__syncthreads();
	colvel.x = sdata[0];

	__syncthreads();
	sdata[tid] = colvel.y;
	__syncthreads();
	reduce(blockSize, tid);
	__syncthreads();
	colvel.y = sdata[0];

	__syncthreads();
	sdata[tid] = colvel.z;
	__syncthreads();
	reduce(blockSize, tid);
	__syncthreads();
	colvel.z = sdata[0];

	__syncthreads();
}

__global__ void cuda_dynamics(double *g_idata, const unsigned int numdynamicobjects_align, const double dt)
{
	//extern __shared__ volatile double sdata[];

	const unsigned int tid = threadIdx.x;
	const unsigned int obj = blockIdx.x;
	const unsigned int numdynamicobjects = gridDim.x;
	const unsigned int blockSize = blockDim.x;

	sdata[tid] = 0.0; 
	if(tid >= numdynamicobjects) return;

	// read in object properties

	double3 pos = make_double3(g_idata[numdynamicobjects_align*0 + obj],
								g_idata[numdynamicobjects_align*1 + obj],
								g_idata[numdynamicobjects_align*2 + obj]);
	double3 vel = make_double3(g_idata[numdynamicobjects_align*3 + obj],
								g_idata[numdynamicobjects_align*4 + obj],
								g_idata[numdynamicobjects_align*5 + obj]);

	double3 kvel = make_double3(0.0, 0.0, 0.0);

	cuda_collisions(blockSize, tid, obj, kvel, pos, vel, g_idata, numdynamicobjects, numdynamicobjects_align);

	vel.x += kvel.x;
	vel.y += kvel.y;
	vel.z += kvel.z;

	// calculate new positions

	double3 inpos = make_double3(pos.x, pos.y, pos.z);
	
	double3 kpos = make_double3(dt*vel.x,
								dt*vel.y,
								dt*vel.z);
	//double3 kvel = make_double3(0.0, 0.0, 0.0);
	kvel.x = 0.0;
	kvel.y = 0.0;
	kvel.z = 0.0;

	kfunction(blockSize, tid, obj, inpos, kvel, g_idata, numdynamicobjects, numdynamicobjects_align);

	kvel.x *= dt*G;
	kvel.y *= dt*G;
	kvel.z *= dt*G;

	double3 newpos = make_double3(pos.x + kpos.x/6.0, 
								  pos.y + kpos.y/6.0, 
								  pos.z + kpos.z/6.0);
	double3 newvel = make_double3(vel.x + kvel.x/6.0, 
								  vel.y + kvel.y/6.0, 
								  vel.z + kvel.z/6.0);

	inpos.x = pos.x + 0.5*kpos.x;
	inpos.y = pos.y + 0.5*kpos.y;
	inpos.z = pos.z + 0.5*kpos.z;

	kpos.x = dt*(vel.x + 0.5*kvel.x);
	kpos.y = dt*(vel.y + 0.5*kvel.y);
	kpos.z = dt*(vel.z + 0.5*kvel.z);
	kvel.x = 0.0;
	kvel.y = 0.0;
	kvel.z = 0.0;

	kfunction(blockSize, tid, obj, inpos, kvel, g_idata, numdynamicobjects, numdynamicobjects_align);

	kvel.x *= dt*G;
	kvel.y *= dt*G;
	kvel.z *= dt*G;

	newpos.x += kpos.x/3.0;
	newpos.y += kpos.y/3.0;
	newpos.z += kpos.z/3.0;
	newvel.x += kvel.x/3.0;
	newvel.y += kvel.y/3.0;
	newvel.z += kvel.z/3.0;

	inpos.x = pos.x + 0.5*kpos.x;
	inpos.y = pos.y + 0.5*kpos.y;
	inpos.z = pos.z + 0.5*kpos.z;

	kpos.x = dt*(vel.x + 0.5*kvel.x);
	kpos.y = dt*(vel.y + 0.5*kvel.y);
	kpos.z = dt*(vel.z + 0.5*kvel.z);
	kvel.x = 0.0;
	kvel.y = 0.0;
	kvel.z = 0.0;

	kfunction(blockSize, tid, obj, inpos, kvel, g_idata, numdynamicobjects, numdynamicobjects_align);

	kvel.x *= dt*G;
	kvel.y *= dt*G;
	kvel.z *= dt*G;

	newpos.x += kpos.x/3.0;
	newpos.y += kpos.y/3.0;
	newpos.z += kpos.z/3.0;
	newvel.x += kvel.x/3.0;
	newvel.y += kvel.y/3.0;
	newvel.z += kvel.z/3.0;

	inpos.x = pos.x + kpos.x;
	inpos.y = pos.y + kpos.y;
	inpos.z = pos.z + kpos.z;

	kpos.x = dt*(vel.x + kvel.x);
	kpos.y = dt*(vel.y + kvel.y);
	kpos.z = dt*(vel.z + kvel.z);
	kvel.x = 0.0;
	kvel.y = 0.0;
	kvel.z = 0.0;

	kfunction(blockSize, tid, obj, inpos, kvel, g_idata, numdynamicobjects, numdynamicobjects_align);

	kvel.x *= dt*G;
	kvel.y *= dt*G;
	kvel.z *= dt*G;

	newpos.x += kpos.x/6.0;
	newpos.y += kpos.y/6.0;
	newpos.z += kpos.z/6.0;
	newvel.x += kvel.x/6.0;
	newvel.y += kvel.y/6.0;
	newvel.z += kvel.z/6.0;

	// collide based on new object position

	//kvel.x = 0.0;
	//kvel.y = 0.0;
	//kvel.z = 0.0;

	//cuda_collisions(blockSize, tid, obj, kvel, newpos, newvel, g_idata, numdynamicobjects, numdynamicobjects_align);

	// calculate final result and write result for this block to global memory
	if (tid == 0) {
		g_idata[obj + 0*numdynamicobjects_align] = newpos.x;
		g_idata[obj + 1*numdynamicobjects_align] = newpos.y;
		g_idata[obj + 2*numdynamicobjects_align] = newpos.z;
		g_idata[obj + 3*numdynamicobjects_align] = newvel.x;
		g_idata[obj + 4*numdynamicobjects_align] = newvel.y;
		g_idata[obj + 5*numdynamicobjects_align] = newvel.z;
	}
}

double* prev_d_in_out = 0;
//double* prev_d_out = 0;
unsigned int prev_numdynamicobjects_in_out = 0;
//unsigned int prev_numdynamicobjects_out = 0;


double* get_d_in_out(const unsigned int numdynamicobjects){
	if (prev_numdynamicobjects_in_out < numdynamicobjects) {
		hipError_t cudaStatus;
		cudaStatus = hipMalloc(&prev_d_in_out, numdynamicobjects*8*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_in hipMalloc failed!\n");
		}
		prev_numdynamicobjects_in_out = numdynamicobjects;
		on_device = false;
	}
	return prev_d_in_out;
}
//double* get_d_out(const unsigned int numdynamicobjects){
//	if (prev_numdynamicobjects_out < numdynamicobjects){
//		hipError_t cudaStatus;
//		cudaStatus = hipMalloc(&prev_d_out, numdynamicobjects*6*sizeof(double));
//		if (cudaStatus != hipSuccess) {
//			fprintf(stderr, "d_out hipMalloc failed!\n");
//		}
//		prev_numdynamicobjects_out = numdynamicobjects;
//	}
//	return prev_d_out;
//}
void freeallcuda(void){
	hipFree(prev_d_in_out);
    //hipFree(prev_d_out);
	prev_numdynamicobjects_in_out = 0;
	//prev_numdynamicobjects_out = 0;
	on_device = false;
}

void do_dynamics(double* dynamic_in_out, const unsigned int numdynamicobjects, const unsigned int numdynamicobjects_align, const double dt)
{
	if (numdynamicobjects > 0){
		hipError_t cudaStatus;

		double* d_in_out = get_d_in_out(numdynamicobjects_align);
		//double* d_out = get_d_out(numdynamicobjects_align);

		if (! on_device){
			// Copy input vectors from host memory to GPU buffers.
			cudaStatus = hipMemcpy(d_in_out, dynamic_in_out, numdynamicobjects_align*8*sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "host to device hipMemcpy failed!\n");
				goto Error;
			}
			on_device = true;
		}

		int dimBlock = 1;
		
		if (numdynamicobjects >= 512)
			dimBlock = 512;
		else 
			while (dimBlock < numdynamicobjects) 
				dimBlock*=2 ;

		int smemSize = dimBlock*sizeof(double);

		cuda_dynamics<<< numdynamicobjects, dimBlock, smemSize >>>(d_in_out, numdynamicobjects_align, dt);

		//scanf("%d");

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = hipMemcpy(dynamic_in_out, d_in_out, numdynamicobjects_align*3*sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "device to host hipMemcpy failed!\n");
			goto Error;
		}
	}

	return;
Error:
	printf("Dynamics Error\n");
	freeallcuda();
	scanf("%d");
	return;
}